
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void vector_add(int* a, int* b, int* c, int n){
    // calc global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < n){
        c[tid] = a[tid] + b[tid];
    }
}

// init vector of size n, with values between 0 and 99
void maxtrix_init(int* a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 100;
    }
}

void error_check(int* a, int* b, int* c, int n) {
    for (int i = 0; i < n; i++) {
        assert(c[i] == a[i] + b[i]);
    }
}

int main(int argc, char** argv){
    
    int n = 1 << 16; // vector size of 2^16

    int *h_a, *h_b, *h_c; // host vector pointers
    int *d_a, *d_b, *d_c; // device vector pointers
    size_t bytes = sizeof(int) * n; // allocation size for vectors

    // allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // initialize vectors a and b
    maxtrix_init(h_a, n);
    maxtrix_init(h_b, n);

    // copy vectors a and b to device memory (gpu) from host memory (cpu)
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice); // inside of the device memory
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    // pointer (d_a) put content of h_a of size bytes

    // Thread block size
    int NUM_THREADS = 512;

    // Grid size
    int NUM_BLOCKS = (int) ceil(n / NUM_THREADS); // we want a single thread calculating each element of vector addition

    // launch kernel
    vector_add<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

    // copy results from device memory (gpu) to host memory (cpu)
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // error checking
    error_check(h_a, h_b, h_c, n);

    // free memory
    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("DONE!\n");
    return 0;
}